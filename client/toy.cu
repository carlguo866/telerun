
#include <hip/hip_runtime.h>
#include <stdio.h>

// CUDA Kernel for vector addition
__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

int main(void)
{
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the host input vectors
    for(int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    // Allocate the device input vector B
    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);

    // Allocate the device output vector C
    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipDeviceSynchronize();

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    // cudaDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.
    hipDeviceReset();

    return 0;
}
